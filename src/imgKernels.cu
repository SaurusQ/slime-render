#include "hip/hip_runtime.h"
#include "imgKernels.cuh"


__global__ void k_convolution(RGB* imgPtr, RGB* imgPadPtr, int* relativeIdxs, float* kernel, unsigned int kernelValues, unsigned int width)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    imgPtr[x + y * width] = RGB{1.0, 1.0, 1.0};
    
/*
    RGB* iPtr = imgPtr;
    RGB* iPadPtr = imgPadPtr;

    int idx = x + y * width; 

    float valueR = 0;
    float valueG = 0;
    float valueB = 0;

    for (int i = 0; i < kernelValues; i++)
    {
        valueR += iPadPtr[relativeIdxs[i]].r * kernel[i];
        valueG += iPadPtr[relativeIdxs[i]].g * kernel[i];
        valueB += iPadPtr[relativeIdxs[i]].b * kernel[i];
    }    

    iPtr[idx].r = valueR / kernelValues;
    iPtr[idx].g = valueG / kernelValues;
    iPtr[idx].b = valueB / kernelValues;
    */  
}