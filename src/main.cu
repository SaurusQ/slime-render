#include "kernelHeader.cuh"
#include "image.hpp"

#include <algorithm>
#include <iostream>

int main()
{
    Image img{};

    while (true)
    {
        std::cout << "Image update" << std::endl;
        img.display();
        img.randomize();
    }

    return 0;
}


/*static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

int main()
{
    hipSetDevice(0);

    float* test = new float[2000];
    float* testGPU = nullptr;

    std::fill(test, test + 2000, 0);


    //hipMalloc((void**)&testGPU, 10000 * sizeof(float));
    CHECK(hipMalloc((void**)&testGPU, 2000 * sizeof(float)));

    CHECK(hipMemcpy(testGPU, test, 2000 * sizeof(float), hipMemcpyHostToDevice));

    kernel<<<1, 1>>>(2, 2, 2, testGPU);

    CHECK(hipMemcpy(test, testGPU, 2000 * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(testGPU);
    delete[] test; 

    return 0;
}*/
