#include "kernelHeader.cuh"

#include <algorithm>
#include <iostream>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

int main()
{
    hipSetDevice(0);

    float* test = new float[2000];
    float* testGPU = nullptr;

    //std::fill(test, test + 2000 * sizeof(float), 0);


    //hipMalloc((void**)&testGPU, 10000 * sizeof(float));
    CHECK(hipMalloc((void**)&testGPU, 2000 * sizeof(float)));

    CHECK(hipMemcpy(testGPU, test, 2000 * sizeof(float), hipMemcpyHostToDevice));

    kernel<<<1, 1>>>(2, 2, 2, testGPU);

    CHECK(hipMemcpy(test, testGPU, 2000 * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(testGPU);
    delete[] test; 

    return 0;
}
