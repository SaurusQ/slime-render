#include "hip/hip_runtime.h"
#include "cudaKernels.cuh"

#include <iostream>

#define PI 3.141592653589793f

void kl_updateTrailMap(dim3 grid, dim3 block,
    double deltaTime,
    RGB* imgPtr,
    RGB* imgPadPtr,
    int* relativeIdxs,
    float diffuseDeltaW,
    float evaporateDeltaW,
    unsigned int width,
    unsigned int padWidth,
    unsigned int padding,
    unsigned int padOffset
)
{
    k_updateTrailMap<<<grid, block>>>(deltaTime, imgPtr, imgPadPtr, relativeIdxs, diffuseDeltaW, evaporateDeltaW, width, padWidth, padding, padOffset);
}

__global__ void k_updateTrailMap(
    double deltaTime,
    RGB* imgPtr,
    RGB* imgPadPtr,
    int* relativeIdxs,
    float diffuseDeltaW,
    float evaporateDeltaW,
    unsigned int width,
    unsigned int padWidth,
    unsigned int padding,
    unsigned int padOffset
)
{
    int x = blockIdx.x * 32 + threadIdx.x;
    int y = blockIdx.y * 32 + threadIdx.y;

    int idxPad = padOffset + x + y * padWidth;
    int idx = x + y * width;

    // Diffuse
    float valueR = 0;
    float valueG = 0;
    float valueB = 0;
    for (int i = 0; i < 9; i++) // 3x3 grid
    {
        valueR += imgPadPtr[idxPad + relativeIdxs[i]].r;
        valueG += imgPadPtr[idxPad + relativeIdxs[i]].g;
        valueB += imgPadPtr[idxPad + relativeIdxs[i]].b;
    }    
    float diffusedR = imgPadPtr[idxPad].r * (1 - diffuseDeltaW) + (valueR / 9.0) * (diffuseDeltaW);
    float diffusedG = imgPadPtr[idxPad].g * (1 - diffuseDeltaW) + (valueB / 9.0) * (diffuseDeltaW);
    float diffusedB = imgPadPtr[idxPad].b * (1 - diffuseDeltaW) + (valueG / 9.0) * (diffuseDeltaW);
    // Evaporate
    imgPtr[idx].r = max(0.0, diffusedR - evaporateDeltaW);
    imgPtr[idx].g = max(0.0, diffusedG - evaporateDeltaW);
    imgPtr[idx].b = max(0.0, diffusedB - evaporateDeltaW);
}

void kl_updateAgents(dim3 grid, dim3 block,
    double deltaTime,
    hiprandState* randomState,
    RGB* imgPtr,
    Agent* agents,
    unsigned int nAgents,
    float speed,
    float turnSpeed,
    float sensorAngleSpacing,
    float sensorOffsetDst,
    unsigned int sensorSize,
    float trailDeltaW,
    unsigned int width,
    unsigned int heigth
)
{
    k_updateAgents<<<grid, block>>>(deltaTime, randomState, imgPtr, agents, nAgents, speed, turnSpeed, sensorAngleSpacing, sensorOffsetDst, sensorSize, trailDeltaW, width, heigth);
}

__global__ void k_updateAgents(
    double deltaTime,
    hiprandState* randomState,
    RGB* imgPtr,
    Agent* agents,
    unsigned int nAgents,
    float speed,
    float turnSpeed,
    float sensorAngleSpacing,
    float sensorOffsetDst,
    unsigned int sensorSize,
    float trailDeltaW,
    unsigned int width,
    unsigned int heigth
)
{
    int agentIdx = blockIdx.x * 32 + threadIdx.x;
    if (agentIdx >= nAgents) return;
    Agent* agent = agents + agentIdx;
    
    // Sense and turn
    float wf = sense(*agent,                 0.0, imgPtr, sensorOffsetDst, sensorSize, width, heigth);
    float wl = sense(*agent,  sensorAngleSpacing, imgPtr, sensorOffsetDst, sensorSize, width, heigth);
    float wr = sense(*agent, -sensorAngleSpacing, imgPtr, sensorOffsetDst, sensorSize, width, heigth);
    
    float randomSteer = hiprand_uniform(randomState + threadIdx.x);


    if (wf > wl && wf > wr)
    {
        agent->angle += 0;
    }
    else if (wf < wl && wf < wr)
    {
        agent->angle += (randomSteer - 0.5) * 2 * turnSpeed * deltaTime;
    }
    else if (wr > wl) {
        agent->angle -= randomSteer * turnSpeed * deltaTime;
    }
    else if (wl > wr)
    {
        agent->angle += randomSteer * turnSpeed * deltaTime;
    }

    // Update position
    float2 direction = make_float2(cosf(agent->angle), sinf(agent->angle));
    float2 newPos = make_float2(deltaTime * speed * direction.x + agent->pos.x, deltaTime * speed * direction.y + agent->pos.y);

    if (newPos.x < 0 || newPos.x >= width || newPos.y < 0 || newPos.y >= heigth)
    {
        newPos.x = min(width - 0.01, max(0.0, newPos.x));
        newPos.y = min(heigth - 0.01, max(0.0, newPos.y));
        agent->angle = hiprand_uniform(randomState + threadIdx.x) * 2 * PI;
    }
    else
    {
        int idx = __float2uint_rd(newPos.x) + __float2uint_rd(newPos.y) * width;
        float3 value = make_float3(imgPtr[idx].r, imgPtr[idx].g, imgPtr[idx].b);
        value.x = min(1.0f, value.x + agent->speciesMask.x * trailDeltaW);
        value.y = min(1.0f, value.y + agent->speciesMask.y * trailDeltaW);
        value.z = min(1.0f, value.z + agent->speciesMask.z * trailDeltaW);
        imgPtr[idx] = RGB{value.x, value.y, value.z};
    }
    
    agent->pos = newPos;
}

__device__ float sense(Agent a, float sensorAngleOffset, RGB* imgPtr, float sensorOffsetDst, int sensorSize, unsigned int width, unsigned int heigth)
{
    float sensorAngle = a.angle + sensorAngleOffset;
    float2 sensorDir = make_float2(cosf(sensorAngle), sinf(sensorAngle));
    int2 sensorCentre = make_int2(a.pos.x + sensorDir.x * sensorOffsetDst, a.pos.y + sensorDir.y * sensorOffsetDst);
    
    float sum = 0.0f;

    int senseWeightX = a.speciesMask.x * 2 - 1;
    int senseWeightY = a.speciesMask.y * 2 - 1;
    int senseWeightZ = a.speciesMask.z * 2 - 1;

    for (int ox = -sensorSize; ox <= sensorSize; ox++)
    {
        for (int oy = -sensorSize; oy <= sensorSize; oy++)
        {
            int2 pos = make_int2(sensorCentre.x + ox, sensorCentre.y + oy);

            if (pos.x >= 0 && pos.x < width && pos.y >= 0 && pos.y < heigth)
            {
                int idx = pos.x + pos.y * width;
                sum += imgPtr[idx].r * senseWeightX
                    + imgPtr[idx].g * senseWeightY
                    + imgPtr[idx].b * senseWeightZ;
            }
        }
    }
    return sum;
}

void kl_initCurand32(dim3 grid, dim3 block,
    hiprandState* state
)
{
    k_initCurand32<<<grid, block>>>(state);
}

__global__ void k_initCurand32(
    hiprandState* state
)
{
    int idx = threadIdx.x;
    hiprand_init(clock64(), idx, 0, state + idx);
}