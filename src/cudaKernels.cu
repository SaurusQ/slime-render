#include "hip/hip_runtime.h"
#include "cudaKernels.cuh"
#include <iostream>

void kl_convolution(dim3 grid, dim3 block,
    RGB* imgPtr,
    RGB* imgPadPtr,
    int* relativeIdxs,
    float* kernel,
    unsigned int kernelValues,
    unsigned int width,
    unsigned int padWidth,
    unsigned int padding,
    unsigned int padOffset
)
{
    k_convolution<<<grid, block>>>(imgPtr, imgPadPtr, relativeIdxs, kernel, kernelValues, width, padWidth, padding, padOffset);
}

__global__ void k_convolution(RGB* imgPtr, RGB* imgPadPtr, int* relativeIdxs, float* kernel, unsigned int kernelValues, unsigned int width, unsigned int padWidth, unsigned int padding, unsigned int padOffset)
{
    int x = blockIdx.x * 32 + threadIdx.x;
    int y = blockIdx.y * 32 + threadIdx.y;

    int idxPad = padOffset + x + y * padWidth;
    int idx = x + y * width; 

    //imgPtr[(idx + 1) % (3840 * 2160)] = imgPadPtr[idxPad];

    //imgPtr[idx] = imgPadPtr[(padding * (2 * padding + width) + padding) + x + y * (width + 2 * padding)];

    /*
    unsigned int count = 0;
    for(int i = 0; i < (3840 + 2 * padding) * (2160 + 2 * padding); i++)
    {
        if(imgPadPtr[i].r >= 0.5) count++;
    }

    printf("count: %u", count);
    */
    
    
    float valueR = 0;
    float valueG = 0;
    float valueB = 0;

    for (int i = 0; i < kernelValues; i++)
    {
        valueR += imgPadPtr[idxPad + relativeIdxs[i]].r * kernel[i];
        valueG += imgPadPtr[idxPad + relativeIdxs[i]].g * kernel[i];
        valueB += imgPadPtr[idxPad + relativeIdxs[i]].b * kernel[i];
    }    

    imgPtr[idx].r = valueR;
    imgPtr[idx].g = valueG;
    imgPtr[idx].b = valueB;
    
}