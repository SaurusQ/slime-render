#include "imageKernel.cuh"
#include "imgKernels.cuh"

#include <cuda_gl_interop.h>

#include <iostream>
#include <stdexcept>

#define REQUIRE_CUDA if(!cudaActive_) { std::cout << "cuda not active" << std::endl; return; };

ImageKernel::ImageKernel(const Image& img, unsigned int padding)
    : padding_(padding)
{
    width_              = img.getWidth();
    height_             = img.getHeight();
    bufferSize_         = img.getBufferSize();
    bufferSizePadded_   = img.getPaddedBufferSize(padding_);

    const hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    /*this->checkCudaError(
        hipMallocArray(&imgPadCudaArray_, &channelDesc, width_ + padding_ * 2, height_ + padding_ * 2),
        "hipMallocArray for imgPadCudaArray_"
    );*/
    this->checkCudaError(
        hipMalloc((void**)&imgPadCudaArray_, bufferSizePadded_),
        "hipMalloc image padded"
    );

    this->checkCudaError(
        hipMemset((void*)imgPadCudaArray_, 0, bufferSizePadded_),
        "hipMemset image padded"
    );
    this->loadTexture();
    this->activateCuda();
    this->update(img);
    this->deactivateCuda();

    /*hipChannelFormatDesc channelDesc;

    // Retrieve the channel format description from the hipArray
    hipGetChannelDesc(&channelDesc, imgCudaArray_);

    // Print the fields of the channel format description
    std::cout << "Channel Format Description:" << std::endl;
    std::cout << "x: " << channelDesc.x << std::endl;
    std::cout << "y: " << channelDesc.y << std::endl;
    std::cout << "z: " << channelDesc.z << std::endl;
    std::cout << "w: " << channelDesc.w << std::endl;
    std::cout << "f: " << channelDesc.f << std::endl;
*/
}

ImageKernel::~ImageKernel()
{
    hipFree((void*)imgPadCudaArray_);
    hipGraphicsUnregisterResource(cudaPboResource_);
    glDeleteTextures(1, &texture_);
}

void ImageKernel::activateCuda()
{
    if (cudaPboResource_ != nullptr)
    {
        cudaActive_ = true;
        this->checkCudaError(
            hipGraphicsMapResources(1, &cudaPboResource_, 0),
            "hipGraphicsMapResources"
        );

        size_t cudaPboSize;
        this->checkCudaError(
            //hipGraphicsResourceGetMappedPointer((void**)&imageGPUptr_, &cudaSize, cudaTextureResource_),
            hipGraphicsResourceGetMappedPointer((void**)&imgCudaArray_, &cudaPboSize, cudaPboResource_),
            "hipGraphicsSubResourceGetMappedArray"
        );

        if (cudaPboSize != bufferSize_)
        {
            std::cerr << "Something wrong with buffer sizes: pbo: " << cudaPboSize << " buffer: " << bufferSize_ << std::endl;
        }
    }
}

void ImageKernel::deactivateCuda()
{
    cudaActive_ = false;
    if (cudaPboResource_ != nullptr)
    {
        hipGraphicsUnmapResources(1, &cudaPboResource_, 0);
    }
    hipDeviceSynchronize();
}

void ImageKernel::update(const Image& img)
{
    REQUIRE_CUDA
    if(img.getBufferSize() != bufferSize_)
    {
        // TODO allow different buffer sizes by reallocatig memory
        std::cout << "Different buffer sizes " << bufferSize_ << " : " << img.getBufferSize() << std::endl;
        return;
    }
    this->checkCudaError(
        //hipMemcpy2DToArray(imgCudaArray_, 0, 0, (void*)img.getPtr(), width_ * sizeof(RGB), width_ * sizeof(RGB), height_, hipMemcpyHostToDevice),
        // ((hipArray_t)imageGPUptr_, 0, 0, (void*)img.getPtr(), bufferSize_, hipMemcpyHostToDevice),
        hipMemcpy((void*)imgCudaArray_, (void*)img.getPtr(), bufferSize_, hipMemcpyHostToDevice),
        "hipMemcpy update()"
    );
    hipDeviceSynchronize();
}

void ImageKernel::readBack(const Image& img) const
{
    REQUIRE_CUDA
    /*this->checkCudaError(
        hipMemcpy((void*)img.getPtr(), (void*)imageGPUptr_, bufferSize_, hipMemcpyDeviceToHost),
        "hipMemcpy readback()"
    );*/
}

void ImageKernel::loadTexture()
{
    // Pixel buffer object
    glGenBuffers(1, &pbo_);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, bufferSize_, nullptr, GL_STREAM_DRAW);

    // Texture
    glGenTextures(1, &texture_);
    glBindTexture(GL_TEXTURE_2D, texture_);
    // set basic parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);


    // Create texture data (4-component unsigned byte)
    //glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_);
    //glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, width_, height_, 0, GL_RGB, GL_FLOAT, NULL);
    //glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    // Unbind the texture
    //glBindTexture(GL_TEXTURE_2D, texture_);

    // cuda device pointer from cuda graphics resource
    
    this->checkCudaError(
        hipGraphicsGLRegisterBuffer(&cudaPboResource_, pbo_, cudaGraphicsMapFlagsWriteDiscard),
        "cudaGraphicsGLRegisterBUffer"
    );
    hipDeviceSynchronize();
}

bool ImageKernel::checkCudaError(hipError_t ce, std::string msg) const
{
    bool failure = ce != hipSuccess;
    if(failure)
    {
        msg = std::string("FAIL: ") + msg + std::string(" WHAT: ") + std::string(hipGetErrorString(ce));
        throw std::runtime_error(msg);
    }
    return failure;
}

void ImageKernel::imgToPadded()
{
    REQUIRE_CUDA
    /*
    this->checkCudaError(
        hipMemcpy2DArrayToArray(imgPadCudaArray_, padding_, padding_, imgCudaArray_, 0, 0, width_, height_),
        "hipMemcpy2DArrayToArray imgToPadded()"
    );*/
    this->checkCudaError(
        hipMemcpy2D((void*)(imgPadCudaArray_ + padding_ + (width_ + 2 * padding_) * padding_), (width_ + 2 * padding_) * sizeof(RGB), (void*)imgCudaArray_, width_ * sizeof(RGB), width_ * sizeof(RGB), height_, hipMemcpyDeviceToDevice),
        "hipMemcpy imgToPadded()"
    );
}

void ImageKernel::convolution(unsigned int kernelSize, const std::vector<float>& kernel)
{
    REQUIRE_CUDA
    unsigned int kernelValues = (kernelSize * 2 + 1) * (kernelSize * 2 + 1);
    if(kernelSize > padding_) 
    {
        std::cerr << "Too large kernel size for padding. Kernel: " << kernelSize << " Padding: " << padding_ << std::endl;
        return; 
    }
    if(kernel.size() < kernelValues)
    {
        std::cerr << "Not enough variables on kernel" << std::endl;
        return;
    }
    this->imgToPadded();
    std::vector<int> relativeIdxs;
    int k = kernelSize;
    for (int y = -k; y <= k; y++)
    {
        for (int x = -k; x <= k; x++)
        {
            relativeIdxs.push_back(x + y * static_cast<int>(width_ + 2 * padding_));
        }
    }

    int* relativeIdxsGPUptr = nullptr;
    float* kernelGPUptr = nullptr;
    this->checkCudaError(
        hipMalloc((void**)&relativeIdxsGPUptr, kernelValues * sizeof(int)),
        "hipMalloc relativeIdxsGPUptr"
    );
    this->checkCudaError(
        hipMemcpy((void*)relativeIdxsGPUptr, relativeIdxs.data(), kernelValues * sizeof(int), hipMemcpyHostToDevice),
        "hipMemcpy relativeIdxs" 
    );
    this->checkCudaError(
        hipMalloc((void**)&kernelGPUptr, kernelValues * sizeof(float)),
        "hipMalloc kernelGPUptr"
    );
    this->checkCudaError(
        hipMemcpy((void*)kernelGPUptr, kernel.data(), kernelValues * sizeof(float), hipMemcpyHostToDevice),
        "hipMemcpy kernel" 
    );

    dim3 dimGrid(120, 72);
    dim3 dimBlock(32, 30);
    k_convolution<<<dimGrid, dimBlock>>>((RGB*)imgCudaArray_, (RGB*)imgPadCudaArray_, relativeIdxsGPUptr, kernelGPUptr, kernelValues, width_, padding_);
    this->checkCudaError(hipGetLastError(), "k_convolution");

    hipDeviceSynchronize();
    hipFree(relativeIdxsGPUptr);
    hipFree(kernelGPUptr);
}
