#include "imageKernel.cuh"
#include "imgKernels.cuh"

#include <cuda_gl_interop.h>

#include <iostream>
#include <stdexcept>

ImageKernel::ImageKernel(const Image& img, unsigned int padding)
{
    width_              = img.getWidth();
    height_             = img.getHeight();
    bufferSize_         = img.getBufferSize();
    bufferSizePadded_   = img.getPaddedBufferSize(padding);

    this->loadTexture();
    this->checkCudaError(
        hipMalloc((void**)&imageGPUpaddedPtr_, bufferSizePadded_),
        "cudaMallow padded image buffer"
    );
    this->checkCudaError(
        hipMemset(imageGPUpaddedPtr_, 0, bufferSizePadded_),
        "hipMemset"
    );
    this->update(img);
}

ImageKernel::~ImageKernel()
{
    hipFree(imageGPUptr_);
    hipGraphicsUnregisterResource(*cudaTextureResource_);
    glDeleteTextures(1, &texture_);
}

void ImageKernel::activateCuda()
{
    if (cudaTextureResource_ != nullptr)
    {
        hipGraphicsMapResources(1, cudaTextureResource_);
    }
}

void ImageKernel::deactivateCuda()
{
    if (cudaTextureResource_ != nullptr)
    {
        hipGraphicsUnmapResources(1, cudaTextureResource_);
    }
}

void ImageKernel::update(const Image& img)
{
    if(img.getBufferSize() != bufferSize_)
    {
        // TODO allow different buffer sizes by reallocatig memory
        std::cout << "Different buffer sizes " << bufferSize_ << " : " << img.getBufferSize() << std::endl;
        return;
    }
    this->checkCudaError(
        hipMemcpy((void*)imageGPUptr_, (void*)img.getPtr(), bufferSize_, hipMemcpyHostToDevice),
        "hipMemcpy"
    );
}

void ImageKernel::readBack(const Image& img) const
{
    this->checkCudaError(
        hipMemcpy((void*)img.getPtr(), (void*)imageGPUptr_, bufferSize_, hipMemcpyDeviceToHost),
        "hipMemcpy"
    );
}

void ImageKernel::loadTexture()
{
    glGenTextures(1, &texture_);
    glBindTexture(GL_TEXTURE_2D, texture_);
    // set basic parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Create texture data (4-component unsigned byte)
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, width_, height_, 0, GL_RGB, GL_FLOAT, NULL);
    // Unbind the texture
    glBindTexture(GL_TEXTURE_2D, texture_);

    // Register the texture with cuda
    this->checkCudaError(
        hipGraphicsGLRegisterImage((hipGraphicsResource**)&cudaTextureResource_, texture_, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard),
        "hipGraphicsGLRegisterImage"
    );
    // cuda device pointer from cuda graphics resource
    this->checkCudaError(
        hipGraphicsMapResources(1, cudaTextureResource_),
        "hipGraphicsMapResources"
    );

    size_t cudaSize;
    this->checkCudaError(
        hipGraphicsResourceGetMappedPointer((void**)&imageGPUptr_, &cudaSize, *cudaTextureResource_),
        "hipGraphicsSubResourceGetMappedArray"
    );

    std::cout << "Got cuda size: " << cudaSize << std::endl;
    std::cout << "Actual cuda size: " << bufferSize_ << std::endl;
}

bool ImageKernel::checkCudaError(hipError_t ce, std::string msg) const
{
    bool failure = ce != hipSuccess;
    if(failure)
    {
        msg = std::string("FAIL: ") + msg + std::string(" WHAT: ") + std::string(hipGetErrorString(ce));
        throw std::runtime_error(msg);
    }
    return failure;
}

void ImageKernel::imgToPadded()
{
    //k_imgToPadded<<<TODO>>>(imageGPUptr_, imageGPUpaddedPtr_);
}

void ImageKernel::convolution(unsigned int kernelSize, const std::vector<float>& kernel)
{
    unsigned int kernelValues = (kernelSize * 2 + 1) * (kernelSize * 2 + 1);
    if(kernelSize > padding_) 
    {
        std::cerr << "Too large kernel size for padding. Kernel: " << kernelSize << " Padding: " << padding_ << std::endl;
        return; 
    }
    if(kernel.size() < kernelValues)
    {
        std::cerr << "Not enough variables on kernel" << std::endl;
        return;
    }
    this->imgToPadded();
    std::vector<int> relativeIdxs;
    for (int y = -kernelSize; y <= kernelSize; y++)
    {
        for (int x = -kernelSize; x <= kernelSize; x++)
        {
            relativeIdxs.push_back(x + y * width_);
        }
    }
    int* relativeIdxsGPUptr = nullptr;
    float* kernelGPUptr = nullptr;
    this->checkCudaError(
        hipMalloc((void**)&relativeIdxsGPUptr, kernelValues * sizeof(int)),
        "hipMalloc relativeIdxsGPUptr"
    );
    this->checkCudaError(
        hipMalloc((void**)&kernelGPUptr, kernelValues * sizeof(float)),
        "hipMalloc kernelGPUptr"
    );
    k_convolution<<<height_, width_>>>(imageGPUptr_, imageGPUpaddedPtr_, relativeIdxsGPUptr, kernelGPUptr, kernelValues, width_);
    hipFree(relativeIdxsGPUptr);
    hipFree(kernelGPUptr);
}
