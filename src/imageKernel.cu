#include "imageKernel.cuh"

#include <iostream>

ImageKernel::ImageKernel(const Image& img)
{
    bufferSize_ = img.getBufferSize();
    hipError_t cudaStatus = hipMalloc((void**)&imageGPUptr_, bufferSize_);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }
    this->update(img);
}

ImageKernel::~ImageKernel()
{
    hipFree(imageGPUptr_);
}

void ImageKernel::update(const Image& img)
{
    if(img.getBufferSize() != bufferSize_)
    {
        std::cout << "Different buffer sizes " << bufferSize_ << " : " << img.getBufferSize() << std::endl;
        return;
    }
    hipError_t cudaStatus = hipMemcpy(imageGPUptr_, img.getPtr(), bufferSize_, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }
}