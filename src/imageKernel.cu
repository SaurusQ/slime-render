#include "imageKernel.cuh"
#include "imgKernels.cuh"

#include <cuda_gl_interop.h>

#include <iostream>
#include <stdexcept>

ImageKernel::ImageKernel(const Image& img, unsigned int padding)
    : padding_(padding)
{
    width_              = img.getWidth();
    height_             = img.getHeight();
    bufferSize_         = img.getBufferSize();
    bufferSizePadded_   = img.getPaddedBufferSize(padding_);

    const hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    /*this->checkCudaError(
        hipMallocArray(&imgPadCudaArray_, &channelDesc, width_ + padding_ * 2, height_ + padding_ * 2),
        "hipMallocArray for imgPadCudaArray_"
    );*/
    this->checkCudaError(
        hipMalloc((void**)&imgPadCudaArray_, bufferSizePadded_),
        "hipMalloc image padded"
    );

    this->checkCudaError(
        hipMemset((void*)imgPadCudaArray_, 0, bufferSizePadded_),
        "hipMemset"
    );
    this->loadTexture();
    this->update(img);

    /*hipChannelFormatDesc channelDesc;

    // Retrieve the channel format description from the hipArray
    hipGetChannelDesc(&channelDesc, imgCudaArray_);

    // Print the fields of the channel format description
    std::cout << "Channel Format Description:" << std::endl;
    std::cout << "x: " << channelDesc.x << std::endl;
    std::cout << "y: " << channelDesc.y << std::endl;
    std::cout << "z: " << channelDesc.z << std::endl;
    std::cout << "w: " << channelDesc.w << std::endl;
    std::cout << "f: " << channelDesc.f << std::endl;
*/
}

ImageKernel::~ImageKernel()
{
    hipFree((void*)imgPadCudaArray_);
    hipGraphicsUnregisterResource(cudaPboResource_);
    glDeleteTextures(1, &texture_);
}

void ImageKernel::activateCuda()
{
    if (cudaPboResource_ != nullptr)
    {
        hipGraphicsMapResources(1, &cudaPboResource_);
    }
}

void ImageKernel::deactivateCuda()
{
    if (cudaPboResource_ != nullptr)
    {
        hipGraphicsUnmapResources(1, &cudaPboResource_);
    }
}

void ImageKernel::update(const Image& img)
{
    if(img.getBufferSize() != bufferSize_)
    {
        // TODO allow different buffer sizes by reallocatig memory
        std::cout << "Different buffer sizes " << bufferSize_ << " : " << img.getBufferSize() << std::endl;
        return;
    }
    this->checkCudaError(
        //hipMemcpy2DToArray(imgCudaArray_, 0, 0, (void*)img.getPtr(), width_ * sizeof(RGB), width_ * sizeof(RGB), height_, hipMemcpyHostToDevice),
        // ((hipArray_t)imageGPUptr_, 0, 0, (void*)img.getPtr(), bufferSize_, hipMemcpyHostToDevice),
        hipMemcpy((void*)imgCudaArray_, (void*)img.getPtr(), bufferSize_, hipMemcpyHostToDevice),
        "hipMemcpy update()"
    );
    hipDeviceSynchronize();
}

void ImageKernel::readBack(const Image& img) const
{
    /*this->checkCudaError(
        hipMemcpy((void*)img.getPtr(), (void*)imageGPUptr_, bufferSize_, hipMemcpyDeviceToHost),
        "hipMemcpy readback()"
    );*/
}

void ImageKernel::loadTexture()
{
    // Pixel buffer object
    glGenBuffers(1, &pbo_);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, bufferSize_, nullptr, GL_DYNAMIC_DRAW);

    // Texture
    glGenTextures(1, &texture_);
    glBindTexture(GL_TEXTURE_2D, texture_);
    // set basic parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);


    // Create texture data (4-component unsigned byte)
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo_);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, width_, height_, 0, GL_RGB, GL_FLOAT, NULL);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    // Unbind the texture
    //glBindTexture(GL_TEXTURE_2D, texture_);

    // cuda device pointer from cuda graphics resource
    
    this->checkCudaError(
        hipGraphicsGLRegisterBuffer(&cudaPboResource_, pbo_, cudaGraphicsMapFlagsWriteDiscard),
        "cudaGraphicsGLRegisterBUffer"
    );

    this->checkCudaError(
        hipGraphicsMapResources(1, &cudaPboResource_, 0),
        "hipGraphicsMapResources"
    );

    std::cout << "hipGraphicsSubResourceGetMappedArray" << std::endl;
    size_t cudaPboSize;
    this->checkCudaError(
        //hipGraphicsResourceGetMappedPointer((void**)&imageGPUptr_, &cudaSize, cudaTextureResource_),
        hipGraphicsResourceGetMappedPointer((void**)&imgCudaArray_, &cudaPboSize, cudaPboResource_),
        "hipGraphicsSubResourceGetMappedArray"
    );

    if (cudaPboSize != bufferSize_)
    {
        std::cerr << "Something wrong with buffer sizes: pbo: " << cudaPboSize << " buffer: " << bufferSize_ << std::endl;
    }
    hipDeviceSynchronize();
}

bool ImageKernel::checkCudaError(hipError_t ce, std::string msg) const
{
    bool failure = ce != hipSuccess;
    if(failure)
    {
        msg = std::string("FAIL: ") + msg + std::string(" WHAT: ") + std::string(hipGetErrorString(ce));
        throw std::runtime_error(msg);
    }
    return failure;
}

void ImageKernel::imgToPadded()
{
    /*
    this->checkCudaError(
        hipMemcpy2DArrayToArray(imgPadCudaArray_, padding_, padding_, imgCudaArray_, 0, 0, width_, height_),
        "hipMemcpy2DArrayToArray imgToPadded()"
    );*/
    this->checkCudaError(
        hipMemcpy2D((void*)imgPadCudaArray_, (width_ + 2 * padding_) * sizeof(RGB), imgCudaArray_, width_ * sizeof(RGB), width_, height_, hipMemcpyDeviceToDevice),
        "hipMemcpy imgToPadded()"
    );
}

void ImageKernel::convolution(unsigned int kernelSize, const std::vector<float>& kernel)
{
    unsigned int kernelValues = (kernelSize * 2 + 1) * (kernelSize * 2 + 1);
    if(kernelSize > padding_) 
    {
        std::cerr << "Too large kernel size for padding. Kernel: " << kernelSize << " Padding: " << padding_ << std::endl;
        return; 
    }
    if(kernel.size() < kernelValues)
    {
        std::cerr << "Not enough variables on kernel" << std::endl;
        return;
    }
    this->imgToPadded();
    std::vector<int> relativeIdxs;
    for (int y = -kernelSize; y <= kernelSize; y++)
    {
        for (int x = -kernelSize; x <= kernelSize; x++)
        {
            relativeIdxs.push_back(x + y * width_);
        }
    }
    int* relativeIdxsGPUptr = nullptr;
    float* kernelGPUptr = nullptr;
    this->checkCudaError(
        hipMalloc((void**)&relativeIdxsGPUptr, kernelValues * sizeof(int)),
        "hipMalloc relativeIdxsGPUptr"
    );
    this->checkCudaError(
        hipMalloc((void**)&kernelGPUptr, kernelValues * sizeof(float)),
        "hipMalloc kernelGPUptr"
    );
    std::cout << "convolustion" << std::endl;
    k_convolution<<<height_, width_>>>((RGB*)imgCudaArray_, (RGB*)imgPadCudaArray_, relativeIdxsGPUptr, kernelGPUptr, kernelValues, width_);
    hipDeviceSynchronize();
    hipFree(relativeIdxsGPUptr);
    hipFree(kernelGPUptr);
}
